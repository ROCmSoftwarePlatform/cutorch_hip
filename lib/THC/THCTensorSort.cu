#include "hip/hip_runtime.h"
#include "THCTensorSort.cuh"

// Returns 2^(ceil(lg(n)) from Stanford bit twiddling hacks
unsigned long nextHighestPowerOf2(unsigned long n) {
  n--;
  n |= n >> 1;
  n |= n >> 2;
  n |= n >> 4;
  n |= n >> 8;
  n |= n >> 16;
#ifndef _MSC_VER
  n |= n >> 32;
#endif
  n++;

  return n;
}

void THCudaLongTensor_fillSliceWithIndex(THCState* state,
                                         THCudaLongTensor* t,
                                         int dim) {
  long dims = THCudaLongTensor_nDimension(state, t);
  THArgCheck(dims <= MAX_CUTORCH_DIMS, 2, CUTORCH_DIM_WARNING);

  ptrdiff_t inElements = THCudaLongTensor_nElement(state, t);
  long sliceSize = THCudaLongTensor_size(state, t, dim);
  ptrdiff_t numSlices = inElements / sliceSize;

  dim3 grid;
  if (!THC_getGridFromTiles(numSlices, grid)) {
    THError("Slice to fill with indices is too large");
  }

  long maxThreads =
    THCState_getCurrentDeviceProperties(state)->maxThreadsPerBlock;
  long numThreads = sliceSize;
  if (numThreads > maxThreads) {
    numThreads = maxThreads;
  }

  dim3 block(numThreads);

#define FILL_INDEX(T, DIM)                                       \
  hipLaunchKernel(reinterpret_cast<const void*>(HIP_KERNEL_NAME(fillSliceWithIndex<T), DIM>),                                     \
      grid, block, 0, THCState_getCurrentStream(state),     \
      infoData, infoSizes, infoStrides, infoDims, numSlices, sliceSize, info.strides[collapseDim])

#ifdef CUDA_PATH
  if (TensorUtils<THCudaLongTensor>::canUse32BitIndexMath(state, t)) {
    TensorInfo<long, unsigned int> info =
      getTensorInfo<THCudaLongTensor, unsigned int>(state, t);
    long* infoData = info.data;
    unsigned int* infoSizes = info.dSizes;
    unsigned int* infoStrides = info.dStrides;
    int infoDims = info.dims;
    info.reduceDim(dim);
    int collapseDim = info.collapseDims(dim);
    if (info.isContiguous()) {
      FILL_INDEX(unsigned int, -2);
    } else {
      if (info.dims == 1) {
        FILL_INDEX(unsigned int, 1);
      } else if (info.dims == 2) {
        FILL_INDEX(unsigned int, 2);
      } else {
        FILL_INDEX(unsigned int, -1);
      }
    }
  } else {
    TensorInfo<long, unsigned long> info =
      getTensorInfo<THCudaLongTensor, unsigned long>(state, t);
    info.reduceDim(dim);
    int collapseDim = info.collapseDims(dim);
    long* infoData = info.data;
    unsigned long* infoSizes = info.dSizes;
    unsigned long* infoStrides = info.dStrides;
    int infoDims = info.dims;

    // catch-all implementation
    FILL_INDEX(unsigned long, -1);
  }
#endif

#undef FILL_INDEX

  THCudaCheck(hipGetLastError());
}
