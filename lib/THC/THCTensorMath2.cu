#include "hip/hip_runtime.h"
#include "THCTensorMath.h"
#include "THCGeneral.h"
#include "THCBlas.h"
#include "THCTensorCopy.h"
#include "THCTensorRandom.h"
#include "THCApply.cuh"
#include "THCReduce.cuh"
#include "THCTensorMathReduce.cuh"
#include "THCTensorMathPointwise.cuh"

#include <bolt/amp/functional.h>
#include <bolt/amp/inner_product.h>


struct TensorTPowOp {
  __host__ __device__
  explicit
  TensorTPowOp(float v) : val{v} {}

  __device__ __forceinline__ void operator()(float* out, float* in) const {
    *out = powf(val, *in);
  }

  __device__ __forceinline__ void operator()(float* v) const {
    *v = powf(val, *v);
  }

  __host__ __device__
  ~TensorTPowOp() {}

  float val;
};

inline
void THCudaTensor_tpow(THCState *state, THCudaTensor *self_, float value, THCudaTensor *src)
{
  THAssert(THCudaTensor_checkGPU(state, 2, self_, src));
  if (self_ == src) {
    if (!THC_pointwiseApply1(state, self_, TensorTPowOp(value))) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  } else {
    THCudaTensor_resizeAs(state, self_, src);

    if (!THC_pointwiseApply2(state, self_, src, TensorTPowOp(value))) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  }

  THCudaCheck(hipGetLastError());
}

struct TensorATan2Op {
  __device__ __forceinline__
  void operator()(float* out, float* a, float* b) const {
    *out = atan2f(*a, *b);
  }
};

void THCudaTensor_atan2(THCState *state, THCudaTensor *self_, THCudaTensor *tx, THCudaTensor *ty)
{
  THAssert(THCudaTensor_checkGPU(state, 3, self_, tx, ty));
  THArgCheck(THCudaTensor_nElement(state, tx) ==
             THCudaTensor_nElement(state, ty), 3, "sizes do not match");
  THCudaTensor_resizeAs(state, self_, tx);

  if (!THC_pointwiseApply3(state, self_, tx, ty, TensorATan2Op())) {
    THArgCheck(false, 2, CUTORCH_DIM_WARNING);
  }

  THCudaCheck(hipGetLastError());
}

float THCudaTensor_dist(THCState *state, THCudaTensor *self, THCudaTensor *src, float value)
{
  THAssert(THCudaTensor_checkGPU(state, 2, self, src));
  self = THCudaTensor_newContiguous(state, self);
  ptrdiff_t size = THCudaTensor_nElement(state, self);
  src = THCudaTensor_newContiguous(state, src);

  auto self_data = THCudaTensor_data(state, self);
  auto src_data = THCudaTensor_data(state, src);

  float result = 0.0; bolt::amp::inner_product( // TODO: add localised version.
#if CUDA_VERSION >= 7000
//    thrust::cuda::par.on(THCState_getCurrentStream(state)),
#endif
    self_data, self_data+size, src_data, (float) 0,
    bolt::amp::plus<float>(), TensorDistOp<float>(value));

  THCudaTensor_free(state, src);
  THCudaTensor_free(state, self);

  return pow(result, (float)1.0/value);
}

void THCudaTensor_rand(THCState *state, THCudaTensor *r_, THLongStorage *size)
{
  THAssert(THCudaTensor_checkGPU(state, 1, r_));
  THCudaTensor_resize(state, r_, size, NULL);
  THCudaTensor_uniform(state, r_, 0, 1);
}

void THCudaTensor_randn(THCState *state, THCudaTensor *r_, THLongStorage *size)
{
  THAssert(THCudaTensor_checkGPU(state, 1, r_));
  THCudaTensor_resize(state, r_, size, NULL);
  THCudaTensor_normal(state, r_, 0, 1);
}
